
#include <hip/hip_runtime.h>
/*
*Autor: Walter Martínez Santana
*/

__global__ void choleskyParalelo(float *db, int num){
	int id=threadIdx.x + blockIdx.x*blockDim.x;
	int x=0;
	int inicio=0;
	int k=0, N=num;
	int id1=id+inicio, ids=id,id2;
	int N2 = N;
	int NN=0, KK=0;
	
	while(k < N){
		id1=id+inicio;
		//Checamos si es un elemnto de la diagonal
		if(id1 == inicio){
			db[id1] = sqrt(db[id1]);
		}else //si no es elemento de la diagonal, lo dividimos por el elemento diagonal de su columna
		{
			x=0;
			while(id1 <N2){
			
				while(x<1000)
					x++;
				__syncthreads();	
				db[id1] = db[id1]/db[inicio];
				
				id1 += gridDim.x * blockDim.x;
			__syncthreads();
			}
			//__syncthreads();//hacemos que todos los threads esperen a los que faltan
			
			
		}__syncthreads();
		//id=ids;
		inicio += (N-k); //Preparo el siguiente salto al siguiente elemento diagonal
		
		
		NN = N2; //Empiezo actaulizar valores de las columnas restantes a la actualizada
		KK = k+1;//cada columna posterior tiene 1 elemento menos a la anterior
		
		while(NN < (int)N*(N+1)/2){
			id2=id + NN; // saltamos a la siguiente columna
			while(id2 < NN + (N-KK)){
				db[id2] = db[id2] -db[id + KK]* db[KK];
				 
				id2 += gridDim.x * blockDim.x;
				__syncthreads();
			}
			//__syncthreads();
			
			NN += (N-KK);
			KK++;
		
		}
		//__syncthreads();
		k++; //pasamos a la siguiente columna
		N2 += (N-k); //Siguiente elemento diagonal
		__syncthreads();
		
	
	}	
	
}
