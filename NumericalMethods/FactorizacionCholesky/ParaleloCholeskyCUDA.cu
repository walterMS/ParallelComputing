
/* Autores: 
* Walter Martínez Santana
* José Carlos Castro
*
*Cholesky en Paralelo en CUDA
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

__global__ void multMatriz(float *da, float *db, float *dc, int num){
	float sum=0;
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	while(j<num){
		while(i<num){
			for (unsigned int k = 0; k<num; k++)
				sum += da[i * num + k] * db[k * num + j];
			dc[i*num + j] = (float) sum;
			i += gridDim.y * blockDim.y;
		}
		j+=gridDim.x * blockDim.x;
		i = threadIdx.y + blockIdx.y * blockDim.y;
	}
	
}



__global__ void choleskyParalelo(float *db, int num){
	int id=threadIdx.x + blockIdx.x*blockDim.x;
	int x=0;
	int inicio=0;
	int k=0, N=num;
	int id1=id+inicio, ids=id,id2;
	int N2 = N;
	int NN=0, KK=0;
	
	while(k < N){
		id1=id+inicio;
		//Checamos si es un elemnto de la diagonal
		if(id1 == inicio){
			db[id1] = sqrt(db[id1]);
		}else //si no es elemento de la diagonal, lo dividimos por el elemento diagonal de su columna
		{
			x=0;
			while(id1 <N2){
			
				while(x<1000)
					x++;
				__syncthreads();	
				db[id1] = db[id1]/db[inicio];
				
				id1 += gridDim.x * blockDim.x;
			__syncthreads();
			}
			//__syncthreads();//hacemos que todos los threads esperen a los que faltan
			
			
		}__syncthreads();
		//id=ids;
		inicio += (N-k); //Preparo el siguiente salto al siguiente elemento diagonal
		
		
		NN = N2; //Empiezo actaulizar valores de las columnas restantes a la actualizada
		KK = k+1;//cada columna posterior tiene 1 elemento menos a la anterior
		
		while(NN < (int)N*(N+1)/2){
			id2=id + NN; // saltamos a la siguiente columna
			while(id2 < NN + (N-KK)){
				db[id2] = db[id2] -db[id + KK]* db[KK];
				 
				id2 += gridDim.x * blockDim.x;
				__syncthreads();
			}
			//__syncthreads();
			
			NN += (N-KK);
			KK++;
		
		}
		//__syncthreads();
		k++; //pasamos a la siguiente columna
		N2 += (N-k); //Siguiente elemento diagonal
		__syncthreads();
		
	
	}	
	
}

#define n 5
#define SIZE n*n*sizeof(float)

int main(){

	int N=n,i,j;
	float *A, *B, *C;
	float *da, *db, *dc;
	int m, P=1,U=6;
	srand(time(NULL));
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	
	dim3 dimGrid(16, 16);
	dim3 dimBlock(16, 16);
	
	A=(float *)malloc(SIZE);
	B=(float *)malloc(SIZE);
	C=(float *)malloc(SIZE);
	for(m=0;m<N*N;m++){
		A[m]=(float)P+(int)(((U-P+1.0)*rand())/(RAND_MAX+1.0));
		//B[m]=(float)P+(int)(((U-P+1.0)*rand())/(RAND_MAX+1.0));
		C[m]=(float)0;
	}
	
	//Transpuesta de A
	for( i = 0;i<N;i++)
		for(j=0;j<N;j++)
			B[j + i*N] = A[i + j*N];
	
	hipMalloc((void**)&da, SIZE);
	hipMalloc((void**)&db, SIZE);
	hipMalloc((void**)&dc, SIZE);
	
	hipMemcpy(da,A, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(db,B, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dc,C, SIZE, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	multMatriz<<<dimGrid , dimBlock >>>(da,db,dc,N);
	//cudaThreadSynchronize();
	hipEventRecord(stop,0);
    hipEventSynchronize(stop);
	
	hipMemcpy(C,dc, SIZE, hipMemcpyDeviceToHost);
	
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(B);
	
	
	//Optimizacion de memoria
	//Almacenamos la parte debajo de la diagonal y la diagonal de la matriz
	int nuevoSize = N*(N+1)/2;
	
	j=0;
	int k;
	B=(float *)malloc(nuevoSize*sizeof(float));
	for(m=0;m<N;m++){
		for(k=m;k<N;k++){
				B[j++]=C[m + N*k];
		}
		
	}
	
	//Desplegar nuevo almacenamiento en arreglo unidimensional
	for(m=0;m<nuevoSize;m++)
		printf("%5.0f ",B[m]);
	
	printf("\n\n");
	
	/*
	for(m=0;m<N*N;m++){
		printf("%08.0f",A[m]);
		printf("%c",( (m%N)<(N-1) ) ? ' ':'\n');
		
	}
	printf("\n\n");
	
	for(m=0;m<N*N;m++){
		printf("%08.0f",B[m]);
		printf("%c",( (m%N)<(N-1) ) ? ' ':'\n');
		
	}
	printf("\n\n");
	*/
	int NN;
	NN=n;
	//for(m=0;m<NN*NN;m++){
	//int NN=16;
	for(m=0;m<NN;m++){
		for(k=0;k<NN;k++){
			printf("%05.0f",C[k + m*N]);
			printf("%c",( ((m*N+k)%NN)<(NN-1) ) ? ' ':'\n');
			//printf("%c",( (m%N)<(N-1) ) ? ' ':'\n');
		}
		
		
	}  
	printf("\n\n");
	
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Tiempo %4.6f milseg\n\n",elapsedTime);
	
	
	hipMalloc((void**)&db, nuevoSize*sizeof(float));
	hipMemcpy(db,B, nuevoSize*sizeof(float), hipMemcpyHostToDevice);
	choleskyParalelo<<<1,512>>>(db,n);
	hipMemcpy(B,db, nuevoSize*sizeof(float), hipMemcpyDeviceToHost);
	
	printf("\n\n");
	for(m=0;m<nuevoSize;m++)
		printf("%4.4f ",B[m]);
	
	printf("\n\n");
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	hipFree(db);
	free(B);
	free(C);
	free(A);
	
	return 0;
}
